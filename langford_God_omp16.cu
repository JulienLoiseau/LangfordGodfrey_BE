#include "hip/hip_runtime.h"
/**
 * \file langford_God_omp16.cu
 * \brief Corps des fonctions de résolution GPU et fonction principale de résolution hybride
 * \author J.Loiseau
 * \version 1.0
 * \date 10/03/2015
 *
 */

#include <stdlib.h>
#include <stdio.h>
#include <sys/timeb.h>
#include <omp.h>

#include "fonctions.h"

/** \brief Tableau de G/G en mémoire constante du GPU (<64KB) */
__constant__ char d_Gray[TAILLEGRAY_GPU];
/** \brief Début de la tâche (selon le process MPI) */
__constant__ long long d_debutT;
/** \brief Tableau de G/G de l'HOST (<20Mo pour E5-2650v2) */
char h_Gray[TAILLEGRAY];


/**
 * \fn void consGray()
 * \brief Construit la partie en mémoire de la suite de G/G sur l'HOST
 *
 */
void consGray() {
	char valeur ;
	int debut, pas ;
	int i ;

	for ( valeur=0, debut=0, pas=2;	valeur < NBGRAY	; valeur++, debut=2*debut+1, pas*=2 )
		for ( i=debut ; i<TAILLEGRAY ; i+=pas )
			h_Gray[i] = valeur ;
}

/**
 * \fn void d_initLocale(memlocale_gpu *ml)
 * \brief Initialisation de la mémoire locale au thread
 *
 * \param ml mémoire locale au thread
 * Dans notre cas l'on fixe les valeurs des bits 1 et 2 à 1 (Voir preuve)
 */
__device__ void d_initLocale(memlocale_gpu *ml) {
	int i;
	/* on fixe x1 = x2 = 1 */
	for (i=1 ; i<=2 ; i++)
		ml->evaluation[i] = 1 ;
	LONG_cree( &(ml->sommeTache) ) ;
	LONG_cree( &(ml->produit) ) ;
}

/**
 * \fn void d_initTermes(memlocale_gpu *ml)
 * \brief Initialisation des termes par rapport à la tâche actuelle
 *
 * \param ml mémoire locale au thread
 */
__device__ void d_initTermes(memlocale_gpu *ml) {
	int i, j ;
	boolean onDoitMultiplier ;

	/* 1: les premiers termes : ce sont les sommes de produits deux à deux */
	/* 1a : i=1 : x1.x3 + ... + x(n-1).x(n+1) (on s'arrête en cours de route) */
	ml->termes[1] = 0 ;
	for ( j=1 ; j <= NBCOUL - 1 ; j++ )
		ml->termes[1] += ml->evaluation[j] * ml->evaluation[j+2] ;
	if ( NBCOUL % 2 ) { /* S1 paire */
		ml->termes[1] /= 2 ;
		onDoitMultiplier = (ml->termes[1] != 0) ;
	}
	else onDoitMultiplier = true ;

	/* 1b : pour i>1 : x1.x(1+i+1) + x2.x(2+i+1) + ... + x(2n-i-1)x(2n) */
	for (i=2 ; i <= NBCOUL ; i++) {
		ml->termes[i] = 0 ;
		for ( j=1 ; j <= NBCUBE - i - 1 ; j++ )
			ml->termes[i] += ml->evaluation[j] * ml->evaluation[j+i+1] ;
		if ( i%2 ) { /* Si paire */
			ml->termes[i] /= 2 ;
			onDoitMultiplier = onDoitMultiplier && (ml->termes[i] != 0) ;
		}
		/* else impair : ne peut pas s'annuler */
	}

	/* 2: le signe ici (je compte les negatifs, puis je donne le signe) */
	ml->signe = 0 ;
	for (i=3 ; i < 3 + TAILLETACHE ; i++)
		if (ml->evaluation[i] < 0)
			ml->signe ++ ;
	ml->signe = (ml->signe % 2 == 0) ? 1 : -1 ;

	/* 3: init. de sommeTache est au premier produit (avec le signe) */
	if (onDoitMultiplier) {
		LONG_init_unite( &(ml->sommeTache) , ml->signe) ;
		for (i=1 ; i <= NBCOUL ; i++)
			LONG_multiplie_char( &(ml->sommeTache) , ml->termes[i]) ;
	}
	else
		LONG_raz( &(ml->sommeTache) ) ;
} /* void initTermes(memlocale *ml, memglobale *mg) */


/**
 * \fn void d_initTache(int numero, memlocale_gpu *ml , char * GrayTab)
 * \brief Initialisation de la mémoire locale par rapport à la tâche numero
 *
 * \param numero numéro de la tâche associée
 * \param ml mémoire locale au thread
 * \param GrayTab tableau de gray conservé en mémoire
 *
 * On regarde par rapport à la tâche convertie en binaire quels sont les valeurs des bits dans les positions (evaluation)
 */
__device__ void d_initTache(long long numero, memlocale_gpu *ml , char * GrayTab) {
	long long i ;
	long long v ;

	/* on calcule la tâche, de 3 à 2+NBTACHE */
	v = numero ;
	for ( i=3 ; i < 3 + TAILLETACHE ; i++ ) {
		ml->evaluation[i] = (v%2 == 1) ? -1 : 1 ;
		v /= 2 ;
	}
	/* fin de l'enumeration */
	/* on fixe toutes les valeurs selon Gray à 1 (avant de commencer) */
	for ( i = 3 + TAILLETACHE ; i <= NBCUBE ; i++ )
		ml->evaluation[i] = 1 ;

	/* les termes au debut, et sommeTache init. au produit des termes initiaux */
	d_initTermes(ml) ;

} /* void initTache(int numero, memlocale *ml, memglobale *mg) */

/**
 * \fn void d_modifierCase(int numCase, memlocale_gpu *ml , char * GrayTab)
 * \brief Modification du signe de la case et répercution de la modification
 *
 * \param numCase numéro de la case à modifier dans le tableau evaluation
 * \param ml mémoire locale au thread
 * \param GrayTab tableau de gray conservé en mémoire
 *
 */
__device__ void d_modifierCase(int numCase, memlocale_gpu *ml) {
	int i ;
	boolean onDoitMultiplier = true ;

	int max1, max2 = NBCOUL ;

	ml->evaluation[numCase] *= -1 ;

	/* terme 1 */
	/* x1.x3 + ... + x(n-1).x(n+1) */
	if ( numCase <= NBCOUL - 1 ) /* il y est en tant que membre de gauche ET droite */
	    /* 1 si S1 peut s'annuler ie si n impair (2 sinon) */
		ml->termes[1] += (2 - NBCOUL % 2) * ml->evaluation[numCase] *
							( ml->evaluation[numCase-2] + ml->evaluation[numCase+2] ) ;
	else
	/* y est-il en tant que membre de droite ? */
		/* ok ssi ( numCase >= 3 && numCase <= NBCOUL+1 )
		 * ie ssi ( numCase <= NBCOUL+1 ) car le numCase est une numGray, >= 3
		 *												(les 2 premiers sont fixes)
		 */
		if ( numCase <=NBCOUL + 1 ) /* il y est DONC comme membre de droite */
			ml->termes[1] += (2 - NBCOUL % 2) * ml->evaluation[numCase-2] * ml->evaluation[numCase] ;
	if (NBCOUL % 2)
		onDoitMultiplier = (ml->termes[1] != 0) ;

if (numCase <= NBCOUL) {
	max1 = numCase - 2 ;
	if (numCase >= NBCOUL - 1)
		max2 = 2 * NBCOUL - numCase - 1 ;

	/* TOUS termes */
/* indices pairs : Si ne peut s'annuler */
	for (i=2 ; i <= max1 ; i += 2) {
		/* a la fois membre gauche et membre droit */
		ml->termes[i] += 2 * ml->evaluation[numCase] *
							( ml->evaluation[numCase-i-1] + ml->evaluation[numCase+i+1] ) ;
	}
	for ( ; i <= max2 ; i += 2) {
		/* seulement membre gauche */
		ml->termes[i] += 2 * ml->evaluation[numCase] * ml->evaluation[numCase+i+1] ;
	}
	/* et c'est tout : sinon, il n'est ni membre gauche, ni membre droit */
/* indice impairs : la somme peut s'annuler ; on utilise les moities */
	for (i=3 ; i <= max1 ; i += 2) {
		/* a la fois membre gauche et membre droit */
		ml->termes[i] += ml->evaluation[numCase] *
							( ml->evaluation[numCase-i-1] + ml->evaluation[numCase+i+1] ) ;
		onDoitMultiplier = onDoitMultiplier && (ml->termes[i] != 0) ;
	}
	for ( ; i <= max2 ; i += 2) {
		/* seulement membre gauche */
		ml->termes[i] += ml->evaluation[numCase] * ml->evaluation[numCase+i+1] ;
		onDoitMultiplier = onDoitMultiplier && (ml->termes[i] != 0) ;
	}
	/* et c'est tout : sinon, il n'est ni membre gauche, ni membre droit */
	/* mais il se peut encore que le terme d'avant soit nul */
	for ( ; i <= NBCOUL ; i += 2)
		onDoitMultiplier = onDoitMultiplier && (ml->termes[i] != 0) ;

} else { /* numCase > NBCOUL */
	max1 = 2 * NBCOUL - numCase - 1 ;
	if (numCase <= NBCOUL + 2)
		max2 = numCase - 2 ;
	else if (numCase >= 2 * NBCOUL - 1)
		max1 = 1 ;

	/* voila TOUS LES AUTRES termes */
/* indices pairs : Si ne peut s'annuler */
	for (i=2 ; i <= max1 ; i += 2) {
		/* a la fois membre gauche et membre droit */
		ml->termes[i] += 2 * ml->evaluation[numCase] *	( ml->evaluation[numCase-i-1] + ml->evaluation[numCase+i+1] ) ;
	}
	for ( ; i <= max2 ; i += 2) {
		/* seulement membre droite */
		ml->termes[i] += 2 * ml->evaluation[numCase-i-1] * ml->evaluation[numCase] ;
	}
	/* et c'est tout : sinon, il n'est ni membre gauche, ni membre droit */
/* indice impairs : la somme peut s'annuler ; on utilise les moities */
	for (i=3 ; i <= max1 ; i += 2) {
		/* a la fois membre gauche et membre droit */
		ml->termes[i] += ml->evaluation[numCase] *( ml->evaluation[numCase-i-1] + ml->evaluation[numCase+i+1] ) ;
		onDoitMultiplier = onDoitMultiplier && (ml->termes[i] != 0) ;
	}
	for ( ; i <= max2 ; i += 2) {
		/* seulement membre droite */
		ml->termes[i] += ml->evaluation[numCase-i-1] * ml->evaluation[numCase] ;
		onDoitMultiplier = onDoitMultiplier && (ml->termes[i] != 0) ;
	}
	/* et c'est tout : sinon, il n'est ni membre gauche, ni membre droit */
	/* mais il se peut encore que le terme d'avant soit nul */
	for ( ; i <= NBCOUL ; i += 2)
		onDoitMultiplier = onDoitMultiplier && (ml->termes[i] != 0) ;

}

	/* cumul (produit, avec le bon signe ; puis incidence sur le produit) */
	ml->signe = - ml->signe ;
	if ( onDoitMultiplier ) {
		LONG_init_unite( &(ml->produit) , ml->signe) ;
		for (i=1 ; i <= NBCOUL ; i++)
			LONG_multiplie_char( &(ml->produit) , ml->termes[i]) ;

		LONG_recale( &(ml->produit) ) ;
		LONG_ajoute_LONG( &(ml->sommeTache) , &(ml->produit) ) ;
	}

} /* void modifierCase(int numCase, memlocale *ml, memglobale *mg) */

/**
 * \fn void d_accumulerTache(memlocale_gpu *ml, char * GrayTab)
 * \brief Parcours de l'ensemble de la suite de G/G et modification les cases en conséquence
 *
 * \param ml mémoire locale au thread
 * \param GrayTab tableau de gray conservé en mémoire
 *
 */
__device__ void d_accumulerTache(memlocale_gpu *ml, char * GrayTab) {
	int numCase ;
	int j,	/* indice dans le tableau mg->Gray */
	       	 t ; /* les "tours" (voir grand commentaire ci-dessous) */
	long gg ; /* pour le nombre de tours */

	int g, k ;

	g = NBCUBE - 2 - TAILLETACHE - ENCOMBREMENT_GPU ; /* le G-E */
	if (g < 0) g = 0 ;	/* mais en faisant attention */

	gg = localPow(2,g) ; /* le 2^(G-E) qui fait attention */

	/* traiter les cases normales (dont le n° selon Gray est dans le tableau) */
	for ( j = 0 ; j < TAILLEGRAY_GPU ; j++ ) {
		numCase = DECALAGEGRAY + GrayTab[j] ;
		d_modifierCase(numCase, ml) ;
	}

	for (t=1 ; t<gg ; t++) { /*tours suivants */

		/* traiter la dernière du tour precedent */
		/*	 (dont on doit CALCULER à la main le numCase à modifier) */
		k = ENCOMBREMENT_GPU ;
		j = t ;
		while ( j % (TAILLEGRAY_GPU + 1) == 0 ) { /* TAILLEGRAY_GPU = 2^ENCOMBREMENT_GPU */
			k += ENCOMBREMENT_GPU ;
			j /= (TAILLEGRAY_GPU + 1) ;
		}
		numCase = DECALAGEGRAY + GrayTab[j-1] + k ;
		d_modifierCase(numCase, ml) ;

		/* traiter les suivantes du tour : qui sont dans Gray (modulo ...) */
		for ( j = 0 ; j < TAILLEGRAY_GPU ; j++ ) {
			numCase = DECALAGEGRAY + GrayTab[j] ;
			d_modifierCase(numCase,ml) ;
		}

	} /* for (t=1 ; t<gg ; t++) */

} /* void accumulerTache(memlocale *ml, memglobale *mg) */

/**
 * \fn void d_compterTache(int t, memlocale_gpu *ml, char * GrayTab)
 * \brief Métafonction pour l'initialisation et le calcul d'une tâche
 *
 * \param t numéro de la tâche
 * \param ml mémoire locale au thread
 * \param GrayTab tableau de gray conservé en mémoire
 *
 */
__device__ void d_compterTache(long long t, memlocale_gpu *ml, char * GrayTab) {
	/* 1: initialisations */
	d_initTache(t, ml, GrayTab) ;

	/* 2: compter */
	d_accumulerTache(ml, GrayTab) ;
} /* void compterTache(int t, memlocale *ml, memglobale *mg) */

/**
 * \fn void HandleError( hipError_t err, const char * file, int line)
 * \brief Gestion et affichage des erreurs CUDA pendant l'exécution
 *
 * \param err type de l'erreur
 * \param file fichier dans lequel l'erreur est localisée
 * \param line ligne dans le fichier où est localisée l'erreur
 *
 */
static void HandleError( hipError_t err, const char * file, int line)
{
	if(err != hipSuccess)
	{
		printf("%s dans %s en ligne line %d\n",hipGetErrorString(err),file,line);
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR(err) (HandleError(err,__FILE__,__LINE__))

/**
 * \fn void resolutionTache(LONG * res)
 * \brief Kernel pour la résolution d'une tâche par un thread
 *
 * \param res tableau des LONG en mémoire DEVICE
 * 
 * Chaque thread s'identifie et calcul sa propre tâche.
 * A la sortie il faut recaler et mettre le LONG résultat dans la case correspondante.
 *
 */
__global__ void resolutionTache(LONG * res)
{
	long long x = blockDim.x * blockIdx.x + threadIdx.x;
	long long tache = x+ d_debutT;
	memlocale_gpu mlT;
	d_initLocale(&mlT);
	d_compterTache(tache,&mlT,d_Gray) ;
	LONG_recale( &(mlT.sommeTache) ) ;
	res[x] = mlT.sommeTache;
}

/**
 * \fn LONG resoudre(int id, int size, int pourcGPU)
 * \brief Fonction principale de résolution hybride CPU/GPU
 *
 * \param id Identifiant MPI du process
 * \param size Nombre total de process MPI
 * \param pourcGPU Pourmille du travail à réaliser par le GPU 
 * 
 * \TODO détailler 
 *
 */
LONG resoudre(int id, int size, int pourcGPU) {


	long long nbTaches = NBTACHE/size;
	long long debutT = id*nbTaches;
	long long finT = debutT + nbTaches;


	//printf("%d in %lu => %lu = (%lu - %lu)\n",id,size,nbTaches,debutT,finT);
	//printf("%d -> %lu taches\n",id,nbTaches);
	
	/* Cas du dernier process MPI, si nombre impair, il fini les tâches */
	if(id == size-1)
	{
		finT = NBTACHE;
	}

	long long i;
	LONG sommeGlobale ;
	LONG_cree(&sommeGlobale) ;
	
/*Constructon de la suite de G/G pour le CPU */
	consGray();
	
	LONG * h_res, * d_res;
	int nbBlocks = 0;
	/********************************** partie GPU ****************************/
if(pourcGPU != 0)
{
	/* Répartition des taches */
	nbBlocks = (int)((nbTaches/nbThreads)*(pourcGPU/1000.f));

	dim3 grid(nbBlocks);
	dim3 blocks(nbThreads);
	//printf("%d GPU : (%d,%d)=%lu, CPU : %lu",id,nbBlocks,nbThreads,nbBlocks*nbThreads,nbTaches-nbBlocks*nbThreads);
	//fflush(stdout);
	
	h_res = (LONG *)malloc(sizeof(LONG)*nbBlocks*nbThreads);

	/* Mémoire device */
	HANDLE_ERROR(hipMalloc((void**)&d_res,sizeof(LONG)*nbBlocks*nbThreads));
/* On copie uniquement la partie utile G/G pour le GPU */	
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_Gray),h_Gray,sizeof(char)*TAILLEGRAY_GPU,0,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_debutT),&debutT,sizeof(long long),0,hipMemcpyHostToDevice));	

	/* Résolution GPU */
	resolutionTache <<< grid, blocks >>>(d_res);
}
	/****************************** partie CPU *********************************/
	
if(pourcGPU != 1000)
{
	long long depart = debutT+nbBlocks*nbThreads;

#pragma omp parallel shared(sommeGlobale) default(shared)
	{ /* debut de la zone parallèle */
		//struct timeb tavt, tapt ;
		long long j;
		memlocale ml;
		initLocale(&ml) ;

		/* 1 : chacun travaille de son cote */
#pragma omp for schedule(static)
		for(j=depart ; j < finT ; j++) {
			compterTache(j,&ml,h_Gray) ;
			LONG_recale( &(ml.sommeTache) ) ;
			LONG_ajoute_LONG( &(ml.sommeProc) , &(ml.sommeTache) ) ; /* 2e recalee */
		} /* fin du for parallèle */

		/* 2 : collecte globale des resultats individuels */
		/* 2a : chaque processeur recale sa somme */
		LONG_recale( &(ml.sommeProc) ) ;
		/* 2b : transmission sur la somme gloabale */
		/* attention : accès concurrent */
#pragma omp critical
		LONG_ajoute_LONG( &sommeGlobale, &(ml.sommeProc) ) ;	/* 2e recalee */

	} /* fin de la zone parallèle */
}

if(pourcGPU != 0)
{

	struct timeb tavt, tapt ;
	ftime(&tavt);

	/*******************Récupérer les calculs GPU******************************/
	HANDLE_ERROR(hipMemcpy(h_res,d_res,sizeof(LONG)*nbBlocks*nbThreads,hipMemcpyDeviceToHost));
	ftime(&tapt);

	//printf("%d Attente CPU : %.3f\n",id,(double)((tapt.time*1000+tapt.millitm)-(tavt.time*1000+tavt.millitm)) / 1000 ) ;
	fflush(stdout);
	/* Sommer le tout */
	for(i = 0 ; i < nbBlocks*nbThreads ; ++i)
	{
		LONG_ajoute_LONG( &sommeGlobale, &(h_res[i]) ) ;
	}
}
	LONG_recalePlus( &sommeGlobale ) ;
if(pourcGPU != 0)
	HANDLE_ERROR(hipFree(d_res));
	return sommeGlobale;
} /* main */
